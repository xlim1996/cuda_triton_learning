
#include <hip/hip_runtime.h>
#include <math.h>
//#include <stdlib.h>
#include <stdio.h>


const double EPSILON = 1.0e-10;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

// kernel function to add the elements of two arrays

__global__ void add(const double *x, const double *y, double *z, int N);

// host function to initialize the arrays

__device__ double add_in_device(const double x, const double y);
__device__ void add_in_device(const double x, const double y, double &z);

// 主机函数。
void check(const double *z, const int N);

int main(){

    const int N = 1e4;
    const int M = sizeof(double) * N;

    // #host memory allocation
    double *h_x = new double[N];
    double *h_y = (double*)malloc(M);
    double *h_z = (double*)malloc(M);

    // 初始化主机数据。
    for (int i = 0; i < N; i++){
        h_x[i] = a;
        h_y[i] = b;
    }

    double *d_x, *d_y, *d_z;
    hipMalloc((void**)&d_x, M);
    hipMalloc((void**)&d_y, M);
    hipMalloc((void**)&d_z, M);

    // #copy data to device
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int grid_size = N / block_size +1;
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);

    // 从设备复制数据。
    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    if (h_x) delete[] h_x;
    free(h_y);
    free(h_z);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    return 0;

}

__global__ void add(const double *x, const double *y, double *z, const int N)
{
    // 在主机函数中需要依次对每个元素进行操作，需要使用一个循环。
    // 在设备函数中，因为采用“单指令-多线程”方式，所以可以去掉循环、只要将数组元素索引和线程索引一一对应即可。

    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n > N) return;

    if (n%5 == 0)
    {
        z[n] = add_in_device(x[n], y[n]);
    }
    else
    {
        add_in_device(x[n], y[n], z[n]);
    }
}

__device__ double add_in_device(const double x, const double y)
{
    return x + y;
}

__device__ void add_in_device(const double x, const double y, double &z)
{
    z = x + y;
}

void check(const double *z, const int N)
{
    bool has_error = false;
    for (int i = 0; i < N ;++i)
    {
        if (fabs(z[i] - c) > EPSILON)
        {
            //printf("%d, %f, %f\n", i, z[i], c);
            has_error = true;
        }
    }

    printf("cuda; %s\n", has_error ? "has error" : "no error");
}