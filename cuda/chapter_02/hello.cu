
#include <hip/hip_runtime.h>
#include<cstdio>
using namespace std;

__global__ void helloFromGPU(void) {

    // get the block index
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int bz = blockIdx.z;

    // #get the thread index
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;

    printf("gpu: hello from block (%d, %d, %d) thread (%d, %d, %d)!\n", bx, by, bz, tx, ty, tz);
}

int main(void) {
    printf("Hello World from nvcc!\n");
    const dim3 blockSize(2, 4);
    const dim3 gridSize (2, 3, 4);
    // grid size is 2x3x4 = 24 blocks
    helloFromGPU<<<gridSize, blockSize>>>();
    hipDeviceSynchronize();
    return 0;
}
// output
// gpu: hello from block (1, 0, 2) thread (0, 0, 0)!
// gpu: hello from block (1, 0, 2) thread (1, 0, 0)!
// gpu: hello from block (1, 0, 2) thread (0, 1, 0)!
// gpu: hello from block (1, 0, 2) thread (1, 1, 0)!
// gpu: hello from block (1, 0, 2) thread (0, 2, 0)!
// gpu: hello from block (1, 0, 2) thread (1, 2, 0)!
// gpu: hello from block (1, 0, 2) thread (0, 3, 0)!
// gpu: hello from block (1, 0, 2) thread (1, 3, 0)!
// gpu: hello from block (1, 0, 1) thread (0, 0, 0)!
// gpu: hello from block (1, 0, 1) thread (1, 0, 0)!
// gpu: hello from block (1, 0, 1) thread (0, 1, 0)!
// gpu: hello from block (1, 0, 1) thread (1, 1, 0)!
// gpu: hello from block (1, 0, 1) thread (0, 2, 0)!
// gpu: hello from block (1, 0, 1) thread (1, 2, 0)!
// gpu: hello from block (1, 0, 1) thread (0, 3, 0)!
// gpu: hello from block (1, 0, 1) thread (1, 3, 0)!
// gpu: hello from block (0, 1, 2) thread (0, 0, 0)!
// gpu: hello from block (0, 1, 2) thread (1, 0, 0)!
// gpu: hello from block (0, 1, 2) thread (0, 1, 0)!
// gpu: hello from block (0, 1, 2) thread (1, 1, 0)!
// gpu: hello from block (0, 1, 2) thread (0, 2, 0)!
// gpu: hello from block (0, 1, 2) thread (1, 2, 0)!
// gpu: hello from block (0, 1, 2) thread (0, 3, 0)!
// gpu: hello from block (0, 1, 2) thread (1, 3, 0)!
// gpu: hello from block (0, 1, 1) thread (0, 0, 0)!
// gpu: hello from block (0, 1, 1) thread (1, 0, 0)!
// gpu: hello from block (0, 1, 1) thread (0, 1, 0)!
// gpu: hello from block (0, 1, 1) thread (1, 1, 0)!
// gpu: hello from block (0, 1, 1) thread (0, 2, 0)!
// gpu: hello from block (0, 1, 1) thread (1, 2, 0)!
// gpu: hello from block (0, 1, 1) thread (0, 3, 0)!
// gpu: hello from block (0, 1, 1) thread (1, 3, 0)!
// gpu: hello from block (1, 0, 3) thread (0, 0, 0)!
// gpu: hello from block (1, 0, 3) thread (1, 0, 0)!
// gpu: hello from block (1, 0, 3) thread (0, 1, 0)!
// gpu: hello from block (1, 0, 3) thread (1, 1, 0)!
// gpu: hello from block (1, 0, 3) thread (0, 2, 0)!
// gpu: hello from block (1, 0, 3) thread (1, 2, 0)!
// gpu: hello from block (1, 0, 3) thread (0, 3, 0)!
// gpu: hello from block (1, 0, 3) thread (1, 3, 0)!
// gpu: hello from block (0, 1, 3) thread (0, 0, 0)!
// gpu: hello from block (0, 1, 3) thread (1, 0, 0)!
// gpu: hello from block (0, 1, 3) thread (0, 1, 0)!
// gpu: hello from block (0, 1, 3) thread (1, 1, 0)!
// gpu: hello from block (0, 1, 3) thread (0, 2, 0)!
// gpu: hello from block (0, 1, 3) thread (1, 2, 0)!
// gpu: hello from block (0, 1, 3) thread (0, 3, 0)!
// gpu: hello from block (0, 1, 3) thread (1, 3, 0)!
// gpu: hello from block (1, 1, 2) thread (0, 0, 0)!
// gpu: hello from block (1, 1, 2) thread (1, 0, 0)!
// gpu: hello from block (1, 1, 2) thread (0, 1, 0)!
// gpu: hello from block (1, 1, 2) thread (1, 1, 0)!
// gpu: hello from block (1, 1, 2) thread (0, 2, 0)!
// gpu: hello from block (1, 1, 2) thread (1, 2, 0)!
// gpu: hello from block (1, 1, 2) thread (0, 3, 0)!
// gpu: hello from block (1, 1, 2) thread (1, 3, 0)!
// gpu: hello from block (0, 2, 2) thread (0, 0, 0)!
// gpu: hello from block (0, 2, 2) thread (1, 0, 0)!
// gpu: hello from block (0, 2, 2) thread (0, 1, 0)!
// gpu: hello from block (0, 2, 2) thread (1, 1, 0)!
// gpu: hello from block (0, 2, 2) thread (0, 2, 0)!
// gpu: hello from block (0, 2, 2) thread (1, 2, 0)!
// gpu: hello from block (0, 2, 2) thread (0, 3, 0)!
// gpu: hello from block (0, 2, 2) thread (1, 3, 0)!
// gpu: hello from block (1, 1, 1) thread (0, 0, 0)!
// gpu: hello from block (1, 1, 1) thread (1, 0, 0)!
// gpu: hello from block (1, 1, 1) thread (0, 1, 0)!
// gpu: hello from block (1, 1, 1) thread (1, 1, 0)!
// gpu: hello from block (1, 1, 1) thread (0, 2, 0)!
// gpu: hello from block (1, 1, 1) thread (1, 2, 0)!
// gpu: hello from block (1, 1, 1) thread (0, 3, 0)!
// gpu: hello from block (1, 1, 1) thread (1, 3, 0)!
// gpu: hello from block (0, 2, 1) thread (0, 0, 0)!
// gpu: hello from block (0, 2, 1) thread (1, 0, 0)!
// gpu: hello from block (0, 2, 1) thread (0, 1, 0)!
// gpu: hello from block (0, 2, 1) thread (1, 1, 0)!
// gpu: hello from block (0, 2, 1) thread (0, 2, 0)!
// gpu: hello from block (0, 2, 1) thread (1, 2, 0)!
// gpu: hello from block (0, 2, 1) thread (0, 3, 0)!
// gpu: hello from block (0, 2, 1) thread (1, 3, 0)!
// gpu: hello from block (0, 1, 0) thread (0, 0, 0)!
// gpu: hello from block (0, 1, 0) thread (1, 0, 0)!
// gpu: hello from block (0, 1, 0) thread (0, 1, 0)!
// gpu: hello from block (0, 1, 0) thread (1, 1, 0)!
// gpu: hello from block (0, 1, 0) thread (0, 2, 0)!
// gpu: hello from block (0, 1, 0) thread (1, 2, 0)!
// gpu: hello from block (0, 1, 0) thread (0, 3, 0)!
// gpu: hello from block (0, 1, 0) thread (1, 3, 0)!
// gpu: hello from block (0, 0, 2) thread (0, 0, 0)!
// gpu: hello from block (0, 0, 2) thread (1, 0, 0)!
// gpu: hello from block (0, 0, 2) thread (0, 1, 0)!
// gpu: hello from block (0, 0, 2) thread (1, 1, 0)!
// gpu: hello from block (0, 0, 2) thread (0, 2, 0)!
// gpu: hello from block (0, 0, 2) thread (1, 2, 0)!
// gpu: hello from block (0, 0, 2) thread (0, 3, 0)!
// gpu: hello from block (0, 0, 2) thread (1, 3, 0)!
// gpu: hello from block (1, 2, 2) thread (0, 0, 0)!
// gpu: hello from block (1, 2, 2) thread (1, 0, 0)!
// gpu: hello from block (1, 2, 2) thread (0, 1, 0)!
// gpu: hello from block (1, 2, 2) thread (1, 1, 0)!
// gpu: hello from block (1, 2, 2) thread (0, 2, 0)!
// gpu: hello from block (1, 2, 2) thread (1, 2, 0)!
// gpu: hello from block (1, 2, 2) thread (0, 3, 0)!
// gpu: hello from block (1, 2, 2) thread (1, 3, 0)!
// gpu: hello from block (1, 2, 3) thread (0, 0, 0)!
// gpu: hello from block (1, 2, 3) thread (1, 0, 0)!
// gpu: hello from block (1, 2, 3) thread (0, 1, 0)!
// gpu: hello from block (1, 2, 3) thread (1, 1, 0)!
// gpu: hello from block (1, 2, 3) thread (0, 2, 0)!
// gpu: hello from block (1, 2, 3) thread (1, 2, 0)!
// gpu: hello from block (1, 2, 3) thread (0, 3, 0)!
// gpu: hello from block (1, 2, 3) thread (1, 3, 0)!
// gpu: hello from block (1, 0, 0) thread (0, 0, 0)!
// gpu: hello from block (1, 0, 0) thread (1, 0, 0)!
// gpu: hello from block (1, 0, 0) thread (0, 1, 0)!
// gpu: hello from block (1, 0, 0) thread (1, 1, 0)!
// gpu: hello from block (1, 0, 0) thread (0, 2, 0)!
// gpu: hello from block (1, 0, 0) thread (1, 2, 0)!
// gpu: hello from block (1, 0, 0) thread (0, 3, 0)!
// gpu: hello from block (1, 0, 0) thread (1, 3, 0)!
// gpu: hello from block (0, 0, 1) thread (0, 0, 0)!
// gpu: hello from block (0, 0, 1) thread (1, 0, 0)!
// gpu: hello from block (0, 0, 1) thread (0, 1, 0)!
// gpu: hello from block (0, 0, 1) thread (1, 1, 0)!
// gpu: hello from block (0, 0, 1) thread (0, 2, 0)!
// gpu: hello from block (0, 0, 1) thread (1, 2, 0)!
// gpu: hello from block (0, 0, 1) thread (0, 3, 0)!
// gpu: hello from block (0, 0, 1) thread (1, 3, 0)!
// gpu: hello from block (1, 2, 0) thread (0, 0, 0)!
// gpu: hello from block (1, 2, 0) thread (1, 0, 0)!
// gpu: hello from block (1, 2, 0) thread (0, 1, 0)!
// gpu: hello from block (1, 2, 0) thread (1, 1, 0)!
// gpu: hello from block (1, 2, 0) thread (0, 2, 0)!
// gpu: hello from block (1, 2, 0) thread (1, 2, 0)!
// gpu: hello from block (1, 2, 0) thread (0, 3, 0)!
// gpu: hello from block (1, 2, 0) thread (1, 3, 0)!
// gpu: hello from block (1, 1, 3) thread (0, 0, 0)!
// gpu: hello from block (1, 1, 3) thread (1, 0, 0)!
// gpu: hello from block (1, 1, 3) thread (0, 1, 0)!
// gpu: hello from block (1, 1, 3) thread (1, 1, 0)!
// gpu: hello from block (1, 1, 3) thread (0, 2, 0)!
// gpu: hello from block (1, 1, 3) thread (1, 2, 0)!
// gpu: hello from block (1, 1, 3) thread (0, 3, 0)!
// gpu: hello from block (1, 1, 3) thread (1, 3, 0)!
// gpu: hello from block (0, 2, 3) thread (0, 0, 0)!
// gpu: hello from block (0, 2, 3) thread (1, 0, 0)!
// gpu: hello from block (0, 2, 3) thread (0, 1, 0)!
// gpu: hello from block (0, 2, 3) thread (1, 1, 0)!
// gpu: hello from block (0, 2, 3) thread (0, 2, 0)!
// gpu: hello from block (0, 2, 3) thread (1, 2, 0)!
// gpu: hello from block (0, 2, 3) thread (0, 3, 0)!
// gpu: hello from block (0, 2, 3) thread (1, 3, 0)!
// gpu: hello from block (0, 0, 3) thread (0, 0, 0)!
// gpu: hello from block (0, 0, 3) thread (1, 0, 0)!
// gpu: hello from block (0, 0, 3) thread (0, 1, 0)!
// gpu: hello from block (0, 0, 3) thread (1, 1, 0)!
// gpu: hello from block (0, 0, 3) thread (0, 2, 0)!
// gpu: hello from block (0, 0, 3) thread (1, 2, 0)!
// gpu: hello from block (0, 0, 3) thread (0, 3, 0)!
// gpu: hello from block (0, 0, 3) thread (1, 3, 0)!
// gpu: hello from block (1, 2, 1) thread (0, 0, 0)!
// gpu: hello from block (1, 2, 1) thread (1, 0, 0)!
// gpu: hello from block (1, 2, 1) thread (0, 1, 0)!
// gpu: hello from block (1, 2, 1) thread (1, 1, 0)!
// gpu: hello from block (1, 2, 1) thread (0, 2, 0)!
// gpu: hello from block (1, 2, 1) thread (1, 2, 0)!
// gpu: hello from block (1, 2, 1) thread (0, 3, 0)!
// gpu: hello from block (1, 2, 1) thread (1, 3, 0)!
// gpu: hello from block (1, 1, 0) thread (0, 0, 0)!
// gpu: hello from block (1, 1, 0) thread (1, 0, 0)!
// gpu: hello from block (1, 1, 0) thread (0, 1, 0)!
// gpu: hello from block (1, 1, 0) thread (1, 1, 0)!
// gpu: hello from block (1, 1, 0) thread (0, 2, 0)!
// gpu: hello from block (1, 1, 0) thread (1, 2, 0)!
// gpu: hello from block (1, 1, 0) thread (0, 3, 0)!
// gpu: hello from block (1, 1, 0) thread (1, 3, 0)!
// gpu: hello from block (0, 2, 0) thread (0, 0, 0)!
// gpu: hello from block (0, 2, 0) thread (1, 0, 0)!
// gpu: hello from block (0, 2, 0) thread (0, 1, 0)!
// gpu: hello from block (0, 2, 0) thread (1, 1, 0)!
// gpu: hello from block (0, 2, 0) thread (0, 2, 0)!
// gpu: hello from block (0, 2, 0) thread (1, 2, 0)!
// gpu: hello from block (0, 2, 0) thread (0, 3, 0)!
// gpu: hello from block (0, 2, 0) thread (1, 3, 0)!
// gpu: hello from block (0, 0, 0) thread (0, 0, 0)!
// gpu: hello from block (0, 0, 0) thread (1, 0, 0)!
// gpu: hello from block (0, 0, 0) thread (0, 1, 0)!
// gpu: hello from block (0, 0, 0) thread (1, 1, 0)!
// gpu: hello from block (0, 0, 0) thread (0, 2, 0)!
// gpu: hello from block (0, 0, 0) thread (1, 2, 0)!
// gpu: hello from block (0, 0, 0) thread (0, 3, 0)!
// gpu: hello from block (0, 0, 0) thread (1, 3, 0)!